//======================================================================
/*
  get_block,  assign threads into a 3D block.
    
    input:
      
      nthreads,     number of threads.
      block_limit,  max dimension of the block.
    
    output:
      
      block_size,   size of the block.
      return value, 0 for all threads are assigned to the block;
                    1 for part of the threads are assigned.
    
  get_grid,   assign threads into a 3D grid, which consists of blocks.
    
    input:
      
      nthreads,     number of threads to assign.
      nthreads_max_per_block, max threads per block.
      grid_limit,   max dimension of the grid.
      block_limit,  max dimension of the block in the grid.
      
    output:
      
      grid_size,    size of the grid.
      block_size,   size of the block.
      return value, 0 for all threads are assigned to the grid;
                    1 for part of the threads are assigned.
*/
//======================================================================

//#include <cstddef>
//#include <cmath>
#include "hip/hip_runtime.h"
//#include "cublas_v2.h"

//// head;
//int get_block( const unsigned long nthreads, const dim3 block_limit, dim3 *block_size );
//int get_grid( const unsigned long nthreads, const unsigned int nthreads_max_per_block, const dim3 grid_limit, const dim3 block_limit, dim3 *grid_size, dim3 *block_size );

int get_block( const unsigned long nthreads, const dim3 block_limit, dim3 *block_size )
{
  unsigned long   v, v_xy, v_xyz;
  unsigned int    zlimit, ylimit, z, y, x;
  
  int             block_status(1);
  unsigned long   nthreads_max(0);
  dim3            max_size(0,0,0);
  
  v = (unsigned long)block_limit.x * (unsigned long)block_limit.y * (unsigned long)block_limit.z;
  if ( v > nthreads )
    v = nthreads;
  
  zlimit = ( v - 1 ) / ( (unsigned long)block_limit.x * (unsigned long)block_limit.y ) + 1;
  for ( z = ( ( 1 > zlimit ) ? 1 : zlimit ); z <= block_limit.z; z++ )
  {
    v_xy = v / z;
    ylimit = ( v_xy - 1 ) / (unsigned long)block_limit.x + 1;
    for ( y = ( ( 1 > ylimit ) ? 1 : ylimit ); y <= block_limit.y; y++ )
    {
      x = v_xy / y;
      if ( x < 1 ) continue;
      
      v_xyz = (unsigned long)x * (unsigned long)y * (unsigned long)z;
      if ( v_xyz > nthreads_max )
      {
        nthreads_max = v_xyz;
        max_size.x = x;
        max_size.y = y;
        max_size.z = z;
        
        if ( v_xyz == v )
        {
          block_status = 0;
          break;
        }
      }
    }
    if ( block_status == 0 )
      break;
  }
  
  if ( nthreads_max == nthreads )
    block_status = 0;
  else
    block_status = 1;
  (*block_size) = max_size;
  
  return block_status;
}

int get_grid( const unsigned long nthreads, const unsigned int nthreads_max_per_block, const dim3 grid_limit, const dim3 block_limit, dim3 *grid_size, dim3 *block_size )
{
  unsigned int    nthreads_per_block_limit, nthreads_per_block;
  unsigned long   nblocks;
  
  int             grid_status(1);
  dim3            bdsize(0,0,0), gdsize(0,0,0);
  unsigned int    nt;
  
  nthreads_per_block_limit = block_limit.x * block_limit.y * block_limit.z;
  if ( nthreads_per_block_limit > nthreads_max_per_block )
    nthreads_per_block_limit = nthreads_max_per_block;
  
  for ( nt = nthreads_per_block_limit; nt >= 1; nt-- )
  {
    get_block( nt, block_limit, &bdsize );
    nthreads_per_block = bdsize.x * bdsize.y * bdsize.z;
    if ( ( nthreads % (unsigned long)nthreads_per_block ) != 0 ) continue;
    
    nblocks = nthreads / (unsigned long)nthreads_per_block;
    grid_status = get_block( nblocks, grid_limit, &gdsize );
    if ( grid_status == 0 ) break;
  }
  
  if ( grid_status == 0 )
  {
    (*grid_size)  = gdsize;
    (*block_size) = bdsize;
  }
  
  return grid_status;
}

//#include <cstddef>
//#include <cmath>
//#include "cuda_runtime.h"
//#include "cublas_v2.h"

//// head;
//int get_block( const size_t *nthreads, const dim3 *block_limit, dim3 *block_size );
//int get_grid( const size_t *nthreads, const size_t *nthreads_max_per_block, const dim3 *grid_limit, const dim3 *block_limit, dim3 *grid_size, dim3 *block_size );

//int get_block( const size_t *nthreads, const dim3 *block_limit, dim3 *block_size )
//{
//  size_t  v, x, y, z, v_xy, v_xyz;
//  size_t  zlimit, ylimit;
//  
//  int     block_status(1);
//  size_t  nthreads_max(0);
//  dim3    max_size(0,0,0);
//  
//  v = (*block_limit).x * (*block_limit).y * (*block_limit).z;
//  if ( v > *nthreads )
//    v = *nthreads;
//  
//  zlimit = ( v - 1 ) / ( (*block_limit).x * (*block_limit).y ) + 1;
//  for ( z = ( ( 1 > zlimit ) ? 1 : zlimit ); z <= (*block_limit).z; z++ )
//  {
//    v_xy = v / z;
//    ylimit = ( v_xy - 1 ) / (*block_limit).x + 1;
//    for ( y = ( ( 1 > ylimit ) ? 1 : ylimit ); y <= (*block_limit).y; y++ )
//    {
//      x = v_xy / y;
//      if ( x < 1 ) continue;
//      
//      v_xyz = x * y * z;
//      if ( v_xyz > nthreads_max )
//      {
//        nthreads_max = v_xyz;
//        max_size.x = x;
//        max_size.y = y;
//        max_size.z = z;
//        
//        if ( v_xyz == v )
//        {
//          block_status = 0;
//          break;
//        }
//      }
//    }
//    if ( block_status == 0 )
//      break;
//  }
//  
//  if ( nthreads_max == *nthreads )
//    block_status = 0;
//  else
//    block_status = 1;
//  (*block_size) = max_size;
//  
//  return block_status;
//}

//int get_grid( const size_t *nthreads, const size_t *nthreads_max_per_block, const dim3 *grid_limit, const dim3 *block_limit, dim3 *grid_size, dim3 *block_size )
//{
//  size_t  nthreads_per_block_limit, nthreads_per_block, nblocks;
//  
//  int     grid_status(1);
//  dim3    bdsize(0,0,0), gdsize(0,0,0);
//  size_t  nt;
//  
//  nthreads_per_block_limit = (*block_limit).x * (*block_limit).y * (*block_limit).z;
//  if ( nthreads_per_block_limit > *nthreads_max_per_block )
//    nthreads_per_block_limit = *nthreads_max_per_block;
//  
//  for ( nt = nthreads_per_block_limit; nt >= 1; nt-- )
//  {
//    get_block( &nt, block_limit, &bdsize );
//    nthreads_per_block = bdsize.x * bdsize.y * bdsize.z;
//    if ( ( *nthreads % nthreads_per_block ) != 0 ) continue;
//    
//    nblocks = *nthreads / nthreads_per_block;
//    grid_status = get_block( &nblocks, grid_limit, &gdsize );
//    if ( grid_status == 0 ) break;
//  }
//  
//  if ( grid_status == 0 )
//  {
//    (*grid_size)  = gdsize;
//    (*block_size) = bdsize;
//  }
//  
//  return grid_status;
//}

