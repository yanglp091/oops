#include "hip/hip_runtime.h"
// |w> = |w> + H * |v>;
// H = sum(H_i) + sum(H_ij);
// H_i = h_i * S_(i,xyz) or h_ij * S_(i,xyz) * S_(j,xyz);
// w_med = H_ij * w_med 

#include <iostream>
#include <complex>
#include <cmath>

#include "hip/hip_runtime.h"
#include "hipblas.h"

// global variable used for texture memory optimization;
texture< int2, 1, hipReadModeElementType > texRef;

//======================================================================
// kron_cuda_v1, y = (Ip @ Am @ Iq) * x;
//======================================================================
__global__ void kron_cuda_v1( const unsigned int m, const size_t s, const unsigned int n, const hipDoubleComplex *A, const unsigned int mat_i_idx_idx, const hipDoubleComplex *x, hipDoubleComplex *y )
{
  hipDoubleComplex     res;
  hipDoubleComplex     mid;
  int2                a1, a2;  

  unsigned int        k, x_idx;
  
  unsigned int        sex((unsigned int)s);
  unsigned int        x_line, A_tex_idx, i;
  
  extern __shared__   hipDoubleComplex x_shd[ ];
  
  //====================================================================
  //  determine the correspondence between thread and vector;
  //  k:      element index, unique for each CUDA thread;
  //  x_idx:  element index, unique in vector x;
  //====================================================================
//  k = blockDim.y * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.y;
  
  k = blockIdx.z * gridDim.y + blockIdx.y;
  k = k * gridDim.x + blockIdx.x;
  k = k * blockDim.y + threadIdx.y;
  
  //  idx2 = s * n * ( k / n ) + k % n;
  //  idx2 = ( s - 1 ) * n * ( k / n ) + k;
  //  x_idx = ( sex - 1 ) * n * ( k / n ) + k + n * threadIdx.x;
  x_idx = k / n;
  x_idx = x_idx * (sex - 1) + threadIdx.x;
  x_idx = x_idx * n + k;
  
  x_line = blockDim.x * threadIdx.y;
  
  //====================================================================
  // copy x to the share memory x_shd;
  //====================================================================
  //  x_shd[ threadIdx.x + blockDim.x * threadIdx.y ] = x[ idx2 + n * threadIdx.x];
  x_shd[threadIdx.x + x_line] = x[x_idx];
  __syncthreads();
  
  //====================================================================
  // matrix multiplication using the global memory with texture memory;
  //====================================================================
//  A_tex_idx = (mat_i_idx_idx + threadIdx.x) * 2;
//  a1 = tex1Dfetch(texRef, A_tex_idx);
//  a2 = tex1Dfetch(texRef, A_tex_idx + 1);
//  mid.x = __hiloint2double(a1.y, a1.x);
//  mid.y = __hiloint2double(a2.y, a2.x);
//  res = hipCmul(x[x_idx - n * threadIdx.x], mid);
//  for (i = 1; i < s; i++)
//  {
//    A_tex_idx = (mat_i_idx_idx + threadIdx.x + sex * i) * 2;
//    a1 = tex1Dfetch(texRef, A_tex_idx);
//    a2 = tex1Dfetch(texRef, A_tex_idx + 1);
//    mid.x = __hiloint2double(a1.y, a1.x);
//    mid.y = __hiloint2double(a2.y, a2.x);
//    res = hipCadd(res, hipCmul(x[x_idx - n * (threadIdx.x - i)], mid));
//  }
  
  //====================================================================
  // matrix multiplication using the share memory without texture memory;
  //====================================================================
//  A_tex_idx = mat_i_idx_idx + threadIdx.x;
//  res = hipCmul(x_shd[x_line], A[A_tex_idx]);
//  for (i = 1; i < s; i++)
//  {
//    res = hipCadd(res, hipCmul(x_shd[i + x_line], A[A_tex_idx + sex * i]));
//  }
  
  //====================================================================
  // matrix multiplication using the share memory with texture memory;
  //====================================================================
  A_tex_idx = (mat_i_idx_idx + threadIdx.x) * 2;
  a1 = tex1Dfetch(texRef, A_tex_idx);
  a2 = tex1Dfetch(texRef, A_tex_idx + 1);
  mid.x = __hiloint2double(a1.y, a1.x);
  mid.y = __hiloint2double(a2.y, a2.x);
  res = hipCmul(x_shd[x_line], mid);
  for (i = 1; i < s; i++)
  {
    A_tex_idx = (mat_i_idx_idx + threadIdx.x + sex * i) * 2;
    a1 = tex1Dfetch(texRef, A_tex_idx);
    a2 = tex1Dfetch(texRef, A_tex_idx + 1);
    mid.x = __hiloint2double(a1.y, a1.x);
    mid.y = __hiloint2double(a2.y, a2.x);
    res = hipCadd(res, hipCmul(x_shd[i + x_line], mid));
  }
  
  //====================================================================
  // add to the result;
  //====================================================================
  y[x_idx] = res;
}
//======================================================================

//======================================================================
// kron_cuda_v3, y = y + coeff * (Ip @ Am @ Iq) * x;
//======================================================================
__global__ void kron_cuda_v3( const unsigned int m, const size_t s, const unsigned int n, const hipDoubleComplex *A, const unsigned int mat_i_idx_idx, const hipDoubleComplex *x, hipDoubleComplex *y, const hipDoubleComplex *coeff )
{
  hipDoubleComplex     res;
  hipDoubleComplex     mid;
  int2                a1, a2;  

  unsigned int        k, x_idx;
  
  unsigned int        sex((unsigned int)s);
  unsigned int        x_line, A_tex_idx, i;
  
  extern __shared__   hipDoubleComplex x_shd[ ];
  
  //====================================================================
  //  determine the correspondence between thread and vector;
  //  k:      element index, unique for each CUDA thread;
  //  x_idx:  element index, unique in vector x;
  //====================================================================
  //  k = blockDim.y * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.y;
  k = blockIdx.z * gridDim.y + blockIdx.y;
  k = k * gridDim.x + blockIdx.x;
  k = k * blockDim.y + threadIdx.y;
  
  //  idx2 = s * n * ( k / n ) + k % n;
  //  idx2 = ( s - 1 ) * n * ( k / n ) + k;
  //  x_idx = ( s - 1 ) * n * ( k / n ) + k + n * threadIdx.x;
  x_idx = k / n;
  x_idx = x_idx * (sex - 1) + threadIdx.x;
  x_idx = x_idx * n + k;
  
  x_line = blockDim.x * threadIdx.y;
  
  //====================================================================
  // copy x to the share memory x_shd;
  //====================================================================
  //  x_shd[ threadIdx.x + blockDim.x * threadIdx.y ] = x[ idx2 + n * threadIdx.x];
  x_shd[threadIdx.x + x_line] = x[x_idx];
  __syncthreads();
  
  //====================================================================
  // matrix multiplication using the global memory with texture memory;
  //====================================================================
//  A_tex_idx = (mat_i_idx_idx + threadIdx.x) * 2;
//  a1 = tex1Dfetch(texRef, A_tex_idx);
//  a2 = tex1Dfetch(texRef, A_tex_idx + 1);
//  mid.x = __hiloint2double(a1.y, a1.x);
//  mid.y = __hiloint2double(a2.y, a2.x);
//  res = hipCmul(x[x_idx - n * threadIdx.x], mid);
//  for (i = 1; i < s; i++)
//  {
//    A_tex_idx = (mat_i_idx_idx + threadIdx.x + sex * i) * 2;
//    a1 = tex1Dfetch(texRef, A_tex_idx);
//    a2 = tex1Dfetch(texRef, A_tex_idx + 1);
//    mid.x = __hiloint2double(a1.y, a1.x);
//    mid.y = __hiloint2double(a2.y, a2.x);
//    res = hipCadd(res, hipCmul(x[x_idx - n * (threadIdx.x - i)], mid));
//  }
  
  //====================================================================
  // matrix multiplication using the share memory without texture memory;
  //====================================================================
//  A_tex_idx = mat_i_idx_idx + threadIdx.x;
//  res = hipCmul(x_shd[x_line], A[A_tex_idx]);
//  for (i = 1; i < s; i++)
//  {
//    res = hipCadd(res, hipCmul(x_shd[i + x_line], A[A_tex_idx + sex * i]));
//  }
  
  //====================================================================
  // matrix multiplication using the share memory with texture memory;
  //====================================================================
  A_tex_idx = (mat_i_idx_idx + threadIdx.x) * 2;
  a1 = tex1Dfetch(texRef, A_tex_idx);
  a2 = tex1Dfetch(texRef, A_tex_idx + 1);
  mid.x = __hiloint2double(a1.y, a1.x);
  mid.y = __hiloint2double(a2.y, a2.x);
  res = hipCmul(x_shd[x_line], mid);
  for (i = 1; i < s; i++)
  {
    A_tex_idx = (mat_i_idx_idx + threadIdx.x + sex * i) * 2;
    a1 = tex1Dfetch(texRef, A_tex_idx);
    a2 = tex1Dfetch(texRef, A_tex_idx + 1);
    mid.x = __hiloint2double(a1.y, a1.x);
    mid.y = __hiloint2double(a2.y, a2.x);
    res = hipCadd(res, hipCmul(x_shd[i + x_line], mid));
  }
  
  //====================================================================
  // add to the result;
  //====================================================================
  res = hipCmul(res, *coeff);
  y[x_idx] = hipCadd(y[x_idx], res);
}
//======================================================================

__global__ void vecrzt_kernel( hipDoubleComplex *x )
{
  size_t  idx;
  
  idx = blockDim.x * ( blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z ) + threadIdx.x;
  
  x[ idx ] = make_hipDoubleComplex(0.0,0.0);
}

#include "include/math/get_grid.h"
//======================================================================
// kron_cuda_v4, y = y + coeff * (Ip @ Am @ Iq @ Bn @ Ir) * x;
//======================================================================
__global__ void kron_cuda_v4(const unsigned int p, const unsigned int m, const unsigned int q, const size_t n, const unsigned int r, const hipDoubleComplex *A, const unsigned int A_idx, const unsigned int B_idx, const hipDoubleComplex *coeff, const hipDoubleComplex *x, hipDoubleComplex *y)
{
  extern __shared__   hipDoubleComplex x_shd[ ];
  hipDoubleComplex     res;
  
  unsigned int        k;
  unsigned int        iq, ir, ip, im, in;
  unsigned int        x_idx;
  
  unsigned int        x_line, Aidx, Bidx, A_tex_idx, B_tex_idx;
  unsigned int        nex((unsigned int)n);
  
  hipDoubleComplex     mul;
  hipDoubleComplex     mid;
  int2                a1, a2;
  
  unsigned int        i, j;
  
  //====================================================================
  //  determine the correspondence between thread and vector;
  //  q * r * p = blockDim.y * gridDim.x * gridDim.y * gridDim.z;
  //  0 <= k < qrp;
  //  
  //  get the numbering of the block;
  //  k:            line index, unique for every line of CUDA block;
  //                determine ip, iq, ir;
  //                k = (blockIdx.x + blockIdx.y * gridDim.x 
  //                    + blockIdx.z * gridDim.y * gridDim.x ) 
  //                    * blockDim.y + threadIdx.y;
  //                  = r * q * ip + r * iq + ir;
  //  threadIdx.x:  row index, unique for every column of CUDA block;
  //                determine im, in;
  //                threadIdx.x = n * im + in;
  //  t, s:         perfect shuffle permutation index;
  //                s = P_{q,m}(t), P denotes the permutation;
  //                t = m * iq + im, numbering in share memory;
  //                s = q * im + iq, numbering in global memory;
  //  x_idx:        element index, unique in vector x;
  //                x_idx = ip * q * m * n * r + iqs * m * n * r 
  //                        + ims * n * r + in * r + ir;
  //====================================================================
  
  //  k = threadIdx.y + (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.y * gridDim.x ) * blockDim.y;
  k = blockIdx.z * gridDim.y + blockIdx.y;
  k = k * gridDim.x + blockIdx.x;
  k = k * blockDim.y + threadIdx.y;
  
  //  ir = k % r;
  //  iq = (k / r) % q;
  //  ip = k / (r * q);
//  ip = k / (r * q);
//  ir = k - r * q * ip;
//  iq = ir / r;
//  ir -= r * iq;
  iq = r * q;
  ip = k / iq;
  ir = k - iq * ip;
  iq = ir / r;
  ir -= r * iq;
  
  // threadIdx.x = n * im + in;
  im = threadIdx.x / nex;
  in = threadIdx.x - nex * im; 
  
  //  s = q * im + iq;
  //  iqs = s / m;
  //  ims = s % m;
  //  x_idx = ip * q * m * n * r + iqs * m * n * r + ims * n * r + in * r + ir;
  x_idx = (((ip * m + im) * q + iq) * nex + in) * r + ir;
  
  x_line = blockDim.x * threadIdx.y;
  
  //====================================================================
  // copy x to the share memory x_shd;
  // x_shd is in row-major order, x is in column-major order;
  //====================================================================
  x_shd[threadIdx.x + x_line] = x[x_idx];
  __syncthreads();
  
  //====================================================================
  // y_shd[threadIdx.y, threadIdx.x]  = x_shd[threadIdx.y,:] 
  //                                  * kron(AT, BT)[:,threadIdx.x];
  // AT (or BT) is the transpose of A (or B);
  // row index of x_shd is threadIdx.y;
  // column indexes of AT and BT are "im" and "in";
  // row indexes of A and B are "im" and "in";
  //====================================================================
  Bidx = B_idx + in;
  Aidx = A_idx + im;
  
  res = make_hipDoubleComplex(0.0, 0.0);
  
  //====================================================================
  // matrix multiplication using the global memory with texture memory;
  //====================================================================
//  for (i = 0; i < m; i++)
//  {
//    mul = make_hipDoubleComplex(0.0, 0.0);
//    x_line = ((ip * m + i) * q + iq) * nex;
//    for (j = 0; j < n; j++)
//    {
//      B_tex_idx = (Bidx + nex * j) * 2;
//      // get B[in + n * j];
//      a1 = tex1Dfetch(texRef, B_tex_idx);
//      a2 = tex1Dfetch(texRef, B_tex_idx + 1);
//      mid.x = __hiloint2double(a1.y, a1.x);
//      mid.y = __hiloint2double(a2.y, a2.x);
//      mul = hipCadd(mul, hipCmul(x[(x_line + j) * r + ir], mid));
//    }
//    A_tex_idx = (Aidx + m * i) * 2;
//    // get A[im + m * i];
//    a1 = tex1Dfetch(texRef, A_tex_idx);
//    a2 = tex1Dfetch(texRef, A_tex_idx + 1);
//    mid.x = __hiloint2double(a1.y, a1.x);
//    mid.y = __hiloint2double(a2.y, a2.x);
//    res = hipCadd(res, hipCmul(mul, mid));
//  }
  
  //====================================================================
  // matrix multiplication using the share memory without texture memory;
  //====================================================================
//  for (i = 0; i < m; i++)
//  {
//    mul = make_hipDoubleComplex(0.0, 0.0);
//    ip = n * i + x_line;
//    for (j = 0; j < n; j++)
//    {
//      B_tex_idx = Bidx + nex * j;
//      mul = hipCadd(mul, hipCmul(x_shd[ip + j], A[B_tex_idx]));
//    }
//    A_tex_idx = Aidx + m * i;
//    res = hipCadd(res, hipCmul(mul, A[A_tex_idx]));
//  }
  
  //====================================================================
  // matrix multiplication using the share memory with texture memory;
  //====================================================================
  for (i = 0; i < m; i++)
  {
    mul = make_hipDoubleComplex(0.0, 0.0);
    for (j = 0; j < n; j++)
    {
      B_tex_idx = (Bidx + nex * j) * 2;
      // get B[in + n * j];
      a1 = tex1Dfetch(texRef, B_tex_idx);
      a2 = tex1Dfetch(texRef, B_tex_idx + 1);
      mid.x = __hiloint2double(a1.y, a1.x);
      mid.y = __hiloint2double(a2.y, a2.x);
      mul = hipCadd(mul, hipCmul(x_shd[n * i + j + x_line], mid));
    }
    A_tex_idx = (Aidx + m * i) * 2;
    // get A[im + m * i];
    a1 = tex1Dfetch(texRef, A_tex_idx);
    a2 = tex1Dfetch(texRef, A_tex_idx + 1);
    mid.x = __hiloint2double(a1.y, a1.x);
    mid.y = __hiloint2double(a2.y, a2.x);
    res = hipCadd(res, hipCmul(mul, mid));
  }
  
  //====================================================================
  // add to the result;
  //====================================================================
  res = hipCmul(res, *coeff);
  y[x_idx] = hipCadd(y[x_idx], res);
}
//======================================================================

void hamvec_cuda3( hipblasHandle_t cublas_handle, int nspin, int nTerm, std::complex<double> *coeff_lst_zplx, size_t *nbody_lst, size_t *pos_i_idx, size_t *pos_i_lst, size_t *dim_i_lst, size_t *mat_i_idx, std::complex<double> *dev_mat_i_lst, size_t vlen, size_t *nspin_dim, size_t *nspin_m_lst, size_t *nspin_n_lst, std::complex<double> *dev_v, std::complex<double> *dev_w, std::complex<double> *dev_w_med, std::complex<double> *dev_coeff_lst_zplx, size_t maxThreadsPerBlock, size_t *maxGridSize )
{
/*
  Calculate the action of a Hamiltonian on a state vector.
  The Hamiltonian can be decomposed into many terms, where each term 
  consists of 
  
  input:
    
    cublas_handle,  handle of cublas;
    nspin,          number of bodies;
    nTerm,          number of interactions in the Hamiltonian;
    coeff_lst_zplx, list of the coefficient in each interaction;
    nbody_lst,      list of the number of bodies in each interaction;
    pos_i_idx,      list of the index of the position list of the body 
                    in each interaction;
    pos_i_lst,      list of the positions of bodies in each interaction;
    dim_i_lst,      list of the dimension of operator of each body in 
                    each interaction;
    mat_i_idx,      list of the index of the operator list of the body
                    in each interaction;
    dev_mat_i_lst,  list of the operators of bodies in each interaction; 
    vlen,           dimension of the state vector;
    nspin_dim,      list of the dimension of each body;
    nspin_m_lst,    list of the dimension of first m bodies;
    nspin_n_lst,    list of the dimension of last n bodies;
    dev_v,          input state vector;
    dev_w_med,      intermediate state vector;
    dev_coeff_lst_zplx, the same as "coeff_lst_zplx";
    maxThreadsPerBlock,   max block size for CUDA;
    maxGridSize,    grid size limit for CUDA;
  
  output:
    
    dev_w,          output state vector, after the Hamiltonian acting on
                    the input state;
*/
  
  size_t  nT, nbody, nb;
  size_t  idx, pos_i, dim_i;
  std::complex<double>  coeff;
  
  size_t  m, n;
  
  dim3    grid_dim, block_dim;
  size_t  dimex1, dimex2;
  
  // optimization by texture memory, bind the operator list;
  hipBindTexture( 0, texRef, dev_mat_i_lst );
  
  size_t          idx1, idx2, pos1, pos2, i;
  size_t          p, q, r;
  unsigned long   nthreads;
  unsigned int    nthreads_max_per_block;
  dim3            blim(1024,1024,64);
  dim3            glim(2147483647,65535,65535);
//  unsigned long   bnlimit(256);
  dim3            grid_limit, block_limit, grid_size, block_size;
  //====================================================================
  // initialization of w;
  //====================================================================
  
  // set block size, 2D;
  block_dim.z = 1;
  block_dim.y = 1;
  dimex1 = maxThreadsPerBlock;
  dimex2 = vlen;
  while ( (dimex2 % dimex1) != 0 ) dimex1--;
  block_dim.x = dimex1;
  dimex2 /= dimex1;
  if ( dimex2 <= maxGridSize[0] * maxGridSize[1] * maxGridSize[2] )
  {
    dimex1 = dimex2 / ( maxGridSize[0] * maxGridSize[1] );
    if ( ( dimex2 % ( maxGridSize[0] * maxGridSize[1] ) ) > 0 )
      dimex1++;
    while ( (dimex2 % dimex1) != 0 ) dimex1++;
    grid_dim.z = dimex1;
    dimex2 = dimex2 / grid_dim.z;
    dimex1 = dimex2 / maxGridSize[0];
    if ( ( dimex2 % maxGridSize[0] ) > 0)
      dimex1++;
    while ( (dimex2 % dimex1) != 0 ) dimex1++;
    grid_dim.y = dimex1;
    grid_dim.x = dimex2 / grid_dim.y;
  }
  else
  {
    std::cout << "block number exceeds limit." << std::endl;
    return;
  }
  
  // reset the output vector;
  vecrzt_kernel<<< grid_dim, block_dim >>>( (hipDoubleComplex*)dev_w );
  
  //====================================================================
  // action of the Hamiltonian on the input state;
  //====================================================================
  
//  // obtain grid;
//  m = dim_i_lst[0];
//  n = dim_i_lst[0];
//  grid_limit = glim;
//  block_limit = blim; 
//  nthreads = vlen / (m * n);
//  nthreads_max_per_block = maxThreadsPerBlock / (m * n);
//  block_limit.x = 1;
//  block_limit.z = 1;
//  get_grid(nthreads, nthreads_max_per_block, grid_limit, block_limit, &grid_size, &block_size );
//  block_size.x = m * n;
  
  // loop over each term of the Hamiltonian;
  for ( nT = 0; nT < nTerm; nT++ )
  {
    coeff = coeff_lst_zplx[ nT ];
    nbody = nbody_lst[ nT ];
    
    if ( abs(coeff) == 0 ) continue;
    
    // loop over each body in each term of the Hamiltonian;
    for ( nb = 0; nb < nbody; nb++ )
    {
      idx = pos_i_idx[ nT ] + nbody - 1 - nb;
      pos_i = pos_i_lst[ idx ];
      dim_i = dim_i_lst[ idx ];
      
      m = nspin_m_lst[ pos_i ];
      n = nspin_n_lst[ pos_i ];
      
      // set block size, 2D;
      block_dim.z = 1;
      block_dim.x = dim_i;
      dimex1 = maxThreadsPerBlock / block_dim.x;
      dimex2 = m * n;
      while ( (dimex2 % dimex1) != 0 ) dimex1--;
      block_dim.y = dimex1;
      // set grid size, 1D-3D;
      dimex2 /= dimex1;// grid size is m * n / block_dim.y;
      if ( dimex2 <= maxGridSize[0] * maxGridSize[1] * maxGridSize[2] )
      {
        dimex1 = dimex2 / ( maxGridSize[0] * maxGridSize[1] );
        if ( ( dimex2 % ( maxGridSize[0] * maxGridSize[1] ) ) > 0 )
          dimex1++;
        while ( (dimex2 % dimex1) != 0 ) dimex1++;
        grid_dim.z = dimex1;
        dimex2 = dimex2 / grid_dim.z;
        dimex1 = dimex2 / maxGridSize[0];
        if ( ( dimex2 % maxGridSize[0] ) > 0)
          dimex1++;
        while ( (dimex2 % dimex1) != 0 ) dimex1++;
        grid_dim.y = dimex1;
        grid_dim.x = dimex2 / grid_dim.y;
      }
      else
      {
        std::cout << "block number exceeds limit." << std::endl;
        return;
      }
      
      if ( nbody == 1 )
      {
//        // for debug;
//        std::cout << "run for nbody == 1. " << std::endl;
        
        kron_cuda_v3<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_v, (hipDoubleComplex*)dev_w, (hipDoubleComplex*)&dev_coeff_lst_zplx[nT] );
      }
      else if (nbody == 2)
      {
        // new 2to1 mat-vec;
        idx2 = pos_i_idx[ nT ] + nbody - 1;
        idx1 = idx2 - 1;
        n = dim_i_lst[ idx2 ];
        m = dim_i_lst[ idx1 ];
        pos2 = pos_i_lst[ idx2 ];
        pos1 = pos_i_lst[ idx1 ];
        
        p = nspin_m_lst[ pos1 ];
        r = nspin_n_lst[ pos2 ];
        q = 1;
        for (i = ( pos1 < pos2 ? pos1 : pos2) + 1; i < ( pos1 < pos2 ? pos2 : pos1); i++)
          q *= nspin_dim[i];
        
        // obtain grid;
        grid_limit = glim;
        block_limit = blim; 
        nthreads = vlen / (m * n);
        nthreads_max_per_block = maxThreadsPerBlock / (m * n);
        block_limit.x = 1;
        block_limit.z = 1;
        get_grid(nthreads, nthreads_max_per_block, grid_limit, block_limit, &grid_size, &block_size );
        block_size.x = m * n;
        
//        // for debug;
//        std::cout << "run for nbody == 2. " << std::endl;
//        std::cout << "p, m, q, n, r = " << p << ", " << m << ", " << q << ", " << n << ", " << r << ", " << std::endl;
//        std::cout << "grid_size.xyz = " << grid_size.x << ", " << grid_size.y << ", " << grid_size.z << std::endl;
//        std::cout << "block_size.xyz = " << block_size.x << ", " << block_size.y << ", " << block_size.z << std::endl;
////        return;
//        std::cout << "A mat idx = " << mat_i_idx[idx1] << std::endl;
//        std::cout << "B mat idx = " << mat_i_idx[idx2] << std::endl;
//        std::complex<double>  *matA, *matB;
//        hipError_t           cuda_status;
//        int i;
//        matA = new std::complex<double> [4];
//        matB = new std::complex<double> [4];
//        cuda_status = hipMemcpy( matA, &dev_mat_i_lst[mat_i_idx[idx1]], ( 4 * sizeof( matA[0] ) ), hipMemcpyDeviceToHost );
//        if (cuda_status != hipSuccess)
//          std::cout << "matA memcpy failed!" << std::endl;
//        cuda_status = hipMemcpy( matB, &dev_mat_i_lst[mat_i_idx[idx2]], ( 4 * sizeof( matB[0] ) ), hipMemcpyDeviceToHost );
//        if (cuda_status != hipSuccess)
//          std::cout << "matB memcpy failed!" << std::endl;
//        std::cout << "matA: " << std::endl;
//        for (i = 0; i < 4; i++)
//          std::cout << i << ", " << matA[i] << std::endl;
//        std::cout << "matB: " << std::endl;
//        for (i = 0; i < 4; i++)
//          std::cout << i << ", " << matB[i] << std::endl;
//        delete[] matA;
//        delete[] matB;
        
        // launch kernel;
        kron_cuda_v4<<< grid_size, block_size, block_size.x * block_size.y * sizeof(std::complex<double>) >>>( p, m, q, n, r, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx1], mat_i_idx[idx2], (hipDoubleComplex*)&dev_coeff_lst_zplx[nT], (hipDoubleComplex*)dev_v, (hipDoubleComplex*)dev_w );
        break;
      }
      else
      {
        // for debug;
        std::cout << "nbody run! skip calculation!" << std::endl;
        return;
        
        if ( nb == 0 )
          kron_cuda_v1<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_v, (hipDoubleComplex*)dev_w_med );
        else if ( nb == nbody - 1 )
          kron_cuda_v3<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_w_med, (hipDoubleComplex*)dev_w, (hipDoubleComplex*)&dev_coeff_lst_zplx[nT] );
        else
        {
          std::cout << "error using, nb should be 1 or 2. " << std::endl;
          return;
//          kron_cuda_v2<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_w_med );
//          kron_cuda_v1<<< grid_dim, block_dim, block_dim.x * block_dim.y * sizeof(std::complex<double>) >>>( m, dim_i, n, (hipDoubleComplex*)dev_mat_i_lst, mat_i_idx[idx], (hipDoubleComplex*)dev_w_med, (hipDoubleComplex*)dev_w_med );
        }
      }
      
    }
    
  }
  
  // optimization by texture memory, release the bind;
  hipUnbindTexture( texRef );
  
//  // for debug;
//  std::cout << "grid_size.xyz = " << grid_size.x << ", " << grid_size.y << ", " << grid_size.z << std::endl;
//  std::cout << "block_size.xyz = " << block_size.x << ", " << block_size.y << ", " << block_size.z << std::endl;
  
}

//======================================================================
// interface for FORTRAN;
//======================================================================

// declaration of function handle;
#define HAMVEC_CUDA3       hamvec_cuda3_
#define HAMVEC_CUDA3_INIT  hamvec_cuda3_init_
#define HAMVEC_CUDA3_TERM  hamvec_cuda3_term_

// delcaration of function;
#if defined(__cplusplus)
extern "C" {
#endif /* __cplusplus */

void HAMVEC_CUDA3( size_t *cublas_handle_ptr, int *nspin_ptr, int *nTerm_ptr, std::complex<double> *coeff_lst_zplx_ptr, size_t *nbody_lst_ptr, size_t *pos_i_idx_ptr, size_t *pos_i_lst_ptr, size_t *dim_i_lst_ptr, size_t *mat_i_idx_ptr, size_t *dev_mat_i_lst_ptr, size_t *ham_dim_ptr, size_t *nspin_dim_ptr, size_t *nspin_m_lst_ptr, size_t *nspin_n_lst_ptr, size_t *dev_v_ptr, size_t *dev_w_ptr, size_t *dev_w_med_ptr, size_t *dev_coeff_lst_zplx_ptr, size_t *maxThreadsPerBlock_ptr, size_t *maxGridSize_ptr );

void HAMVEC_CUDA3_INIT( size_t *cublas_handle_ptr );

void HAMVEC_CUDA3_TERM( size_t *cublas_handle_ptr );

#if defined(__cplusplus)
}
#endif /* __cplusplus */

// interface of function;
void HAMVEC_CUDA3( size_t *cublas_handle_ptr, int *nspin_ptr, int *nTerm_ptr, std::complex<double> *coeff_lst_zplx_ptr, size_t *nbody_lst_ptr, size_t *pos_i_idx_ptr, size_t *pos_i_lst_ptr, size_t *dim_i_lst_ptr, size_t *mat_i_idx_ptr, size_t *dev_mat_i_lst_ptr, size_t *ham_dim_ptr, size_t *nspin_dim_ptr, size_t *nspin_m_lst_ptr, size_t *nspin_n_lst_ptr, size_t *dev_v_ptr, size_t *dev_w_ptr, size_t *dev_w_med_ptr, size_t *dev_coeff_lst_zplx_ptr, size_t *maxThreadsPerBlock_ptr, size_t *maxGridSize_ptr )
{
  hipblasHandle_t        cublas_handle   = (hipblasHandle_t)*cublas_handle_ptr;
  int                   nspin           = *nspin_ptr;
  int                   nTerm           = *nTerm_ptr;
  std::complex<double>  *coeff_lst_zplx = coeff_lst_zplx_ptr;
  size_t                *nbody_lst      = nbody_lst_ptr;
  size_t                *pos_i_idx      = pos_i_idx_ptr;
  size_t                *pos_i_lst      = pos_i_lst_ptr;
  size_t                *dim_i_lst      = dim_i_lst_ptr;
  size_t                *mat_i_idx      = mat_i_idx_ptr;
  std::complex<double>  *dev_mat_i_lst  = (std::complex<double>*)(*dev_mat_i_lst_ptr);
  size_t                vlen            = *ham_dim_ptr;
  size_t                *nspin_dim      = nspin_dim_ptr;
  size_t                *nspin_m_lst    = nspin_m_lst_ptr;
  size_t                *nspin_n_lst    = nspin_n_lst_ptr;
  std::complex<double>  *dev_v          = (std::complex<double>*)(*dev_v_ptr);
  std::complex<double>  *dev_w          = (std::complex<double>*)(*dev_w_ptr);
  std::complex<double>  *dev_w_med      = (std::complex<double>*)(*dev_w_med_ptr);
  std::complex<double>  *dev_coeff_lst_zplx = (std::complex<double>*)(*dev_coeff_lst_zplx_ptr);
  size_t            maxThreadsPerBlock  = *maxThreadsPerBlock_ptr;
  size_t                *maxGridSize    = maxGridSize_ptr;
  
  hamvec_cuda3( cublas_handle, nspin, nTerm, coeff_lst_zplx, nbody_lst, pos_i_idx, pos_i_lst, dim_i_lst, mat_i_idx, dev_mat_i_lst, vlen, nspin_dim, nspin_m_lst, nspin_n_lst, dev_v, dev_w, dev_w_med, dev_coeff_lst_zplx, maxThreadsPerBlock, maxGridSize );
}

void HAMVEC_CUDA3_INIT( size_t *cublas_handle_ptr )
{
  // initialization of cublas handle for FORTRAN;
  hipblasHandle_t cublas_handle;
  hipblasCreate( &cublas_handle );
  *cublas_handle_ptr = (size_t)cublas_handle;
}

void HAMVEC_CUDA3_TERM( size_t *cublas_handle_ptr )
{
  // termination of cublas handle for FORTRAN;
  hipblasHandle_t cublas_handle;
  cublas_handle = (hipblasHandle_t)*cublas_handle_ptr;
  hipblasDestroy( cublas_handle );
}

